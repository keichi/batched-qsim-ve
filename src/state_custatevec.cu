#include "hip/hip_runtime.h"
// vim: set ft=cuda:
#include <cmath>
#include <random>
#include <stdexcept>
#include <vector>

#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>
#include <custatevec.h>

#include "state.hpp"

#define HANDLE_ERROR(x)                                                                            \
    {                                                                                              \
        const auto err = x;                                                                        \
        if (err != CUSTATEVEC_STATUS_SUCCESS) {                                                    \
            printf("cuStateVec error \"%s\" at %s:%d\n", custatevecGetErrorString(err), __FILE__,  \
                   __LINE__);                                                                      \
            std::exit(EXIT_FAILURE);                                                               \
        }                                                                                          \
    };

#define HANDLE_CUDA_ERROR(x)                                                                       \
    {                                                                                              \
        const auto err = x;                                                                        \
        if (err != hipSuccess) {                                                                  \
            printf("CUDA Error: \"%s\" at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__);  \
            std::exit(EXIT_FAILURE);                                                               \
        }                                                                                          \
    };

namespace veqsim
{

class State::Impl
{
public:
    Impl(UINT n, UINT batch_size)
        : n_(n), batch_size_(batch_size), mt_engine_(seed_gen_()), dist_(0.0, 1.0)
    {
        HANDLE_CUDA_ERROR(hipMalloc(&state_, batch_size * (1ULL << n) * sizeof(hipDoubleComplex)));
    }

    ~Impl() { HANDLE_CUDA_ERROR(hipFree(state_)); }

    static void initialize() { HANDLE_ERROR(custatevecCreate(&handle_)); }

    static void finalize() { HANDLE_ERROR(custatevecDestroy(handle_)); }

    std::vector<std::complex<double>> get_vector(UINT sample) const
    {
        std::vector<std::complex<double>> sv(1ULL << n_);

        HANDLE_CUDA_ERROR(hipMemcpy(sv.data(), state_ + (1ULL << n_) * sample,
                                     (1ULL << n_) * sizeof(hipDoubleComplex),
                                     hipMemcpyDeviceToHost));

        return sv;
    }

    std::complex<double> amplitude(UINT sample, UINT i)
    {
        hipDoubleComplex c;
        HANDLE_CUDA_ERROR(hipMemcpy(&c, state_ + (1ULL << n_) * sample + i,
                                     sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        return std::complex(hipCreal(c), hipCimag(c));
    }

    double re(UINT sample, UINT i) { return amplitude(sample, i).real(); }

    double im(UINT sample, UINT i) { return amplitude(sample, i).imag(); }

    double get_probability(UINT i) { throw std::runtime_error("Not implemented"); }

    double get_probability(UINT sample, UINT i) { return std::norm(amplitude(sample, i)); }

    std::vector<double> get_probability_batched(UINT i) const
    {
        std::vector<double> probs(batch_size_);
        std::vector<custatevecIndex_t> mask_bit_string(batch_size_);
        std::vector<int32_t> mask_ordering(n_);
        std::iota(mask_ordering.begin(), mask_ordering.end(), 0);

        for (int i = 0; i < n_; i++) {
            mask_ordering[i] = i;
        }

        HANDLE_ERROR(custatevecAbs2SumArrayBatched(
            handle_,                // custatevecHandle_t handle
            state_,                 // const void *batchedSv
            HIP_C_64F,             // hipDataType svDataType
            n_,                     // const uint32_t nIndexBits
            batch_size_,            // const uint32_t nSVs
            1ULL << n_,             // const custatevecIndex_t svStride
            probs.data(),           // double *abs2sumArrays
            1,                      // const custatevecIndex_t abs2sumArrayStride
            nullptr,                // const int32_t *bitOrdering
            0,                      // const uint32_t bitOrderingLen
            mask_bit_string.data(), // const custatevecIndex_t *maskBitStrings
            mask_ordering.data(),   // const int32_t *maskOrdering
            n_                      // const uint32_t maskLen
            ));

        return probs;
    }

    UINT dim() const { return 1ULL << n_; }

    UINT batch_size() const { return batch_size_; }

    void set_zero_state()
    {
        std::vector<hipDoubleComplex> state(batch_size_ * (1ULL << n_));

        for (ITYPE i = 0; i < state.size(); i += (1ULL << n_)) {
            state[i] = make_hipDoubleComplex(1, 0);
        }

        HANDLE_CUDA_ERROR(hipMemcpy(state_, state.data(), state.size() * sizeof(hipDoubleComplex),
                                     hipMemcpyHostToDevice));
    }

    void act_single_qubit_gate(UINT target, hipDoubleComplex matrix[2][2])
    {
        int32_t targets[] = {static_cast<int32_t>(target)};

        // size_t workspace_size;
        // HANDLE_ERROR(custatevecApplyMatrixBatchedGetWorkspaceSize(
        //     handle_, HIP_C_64F, n_, batch_size_, (1ULL << n_),
        //     CUSTATEVEC_MATRIX_MAP_TYPE_BROADCAST, nullptr, matrix, HIP_C_64F,
        //     CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, 1, 1, 0, CUSTATEVEC_COMPUTE_64F,
        //     &workspace_size));

        HANDLE_ERROR(custatevecApplyMatrixBatched(
            handle_,                              // custatevecHandle_t handle
            state_,                               // void *batchedSv
            HIP_C_64F,                           // hipDataType svDataType
            n_,                                   // const uint32_t nIndexBits
            batch_size_,                          // const uint32_t nSVs
            1ULL << n_,                           // custatevecIndex_t svStride
            CUSTATEVEC_MATRIX_MAP_TYPE_BROADCAST, // custatevecMatrixMapType_t mapType
            nullptr,                              // const int32_t *matrixIndices
            matrix,                               // const void *matrices
            HIP_C_64F,                           // hipDataType matrixDataType
            CUSTATEVEC_MATRIX_LAYOUT_ROW,         // custatevecMatrixLayout_t layout
            0,                                    // const int32_t adjoint
            1,                                    // const uint32_t nMatrices
            targets,                              // const int32_t *targets
            1,                                    // const uint32_t nTargets
            nullptr,                              // const int32_t *controls
            nullptr,                              // const int32_t *controlBitValues
            0,                                    // const uint32_t nControls
            CUSTATEVEC_COMPUTE_64F,               // custatevecComputeType_t computeType
            nullptr,                              // void *extraWorkspace
            0                                     // size_t extraWorkspaceSizeInBytes
            ));
    }

    void act_two_qubit_gate(UINT control, UINT target, hipDoubleComplex matrix[4][4])
    {
        int32_t targets[] = {static_cast<int32_t>(target), static_cast<int32_t>(control)};

        // size_t workspace_size;
        // HANDLE_ERROR(custatevecApplyMatrixBatchedGetWorkspaceSize(
        //     handle_, HIP_C_64F, n_, batch_size_, (1ULL << n_),
        //     CUSTATEVEC_MATRIX_MAP_TYPE_BROADCAST, nullptr, matrix, HIP_C_64F,
        //     CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, 1, 2, 0, CUSTATEVEC_COMPUTE_64F,
        //     &workspace_size));

        HANDLE_ERROR(custatevecApplyMatrixBatched(
            handle_,                              // custatevecHandle_t handle
            state_,                               // void *batchedSv
            HIP_C_64F,                           // hipDataType svDataType
            n_,                                   // const uint32_t nIndexBits
            batch_size_,                          // const uint32_t nSVs
            1ULL << n_,                           // custatevecIndex_t svStride
            CUSTATEVEC_MATRIX_MAP_TYPE_BROADCAST, // custatevecMatrixMapType_t mapType
            nullptr,                              // const int32_t *matrixIndices
            matrix,                               // const void *matrices
            HIP_C_64F,                           // hipDataType matrixDataType
            CUSTATEVEC_MATRIX_LAYOUT_ROW,         // custatevecMatrixLayout_t layout
            0,                                    // const int32_t adjoint
            1,                                    // const uint32_t nMatrices
            targets,                              // const int32_t *targets
            2,                                    // const uint32_t nTargets
            nullptr,                              // const int32_t *controls
            nullptr,                              // const int32_t *controlBitValues
            0,                                    // const uint32_t nControls
            CUSTATEVEC_COMPUTE_64F,               // custatevecComputeType_t computeType
            nullptr,                              // void *extraWorkspace
            0                                     // size_t extraWorkspaceSizeInBytes
            ));
    }

    void act_x_gate(UINT target)
    {
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0)},
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_y_gate(UINT target)
    {
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, -1)},
            {make_hipDoubleComplex(0, -1), make_hipDoubleComplex(0, 0)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_z_gate(UINT target)
    {
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(-1, 0)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_h_gate(UINT target)
    {
        static double inv_sqrt2 = 1 / std::sqrt(2);
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(inv_sqrt2, 0), make_hipDoubleComplex(inv_sqrt2, 0)},
            {make_hipDoubleComplex(inv_sqrt2, 0), make_hipDoubleComplex(-inv_sqrt2, 0)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_rx_gate(UINT target, double theta)
    {
        double cos_half = std::cos(theta / 2), sin_half = std::sin(theta / 2);
        hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(cos_half, 0), make_hipDoubleComplex(0, -sin_half)},
            {make_hipDoubleComplex(0, -sin_half), make_hipDoubleComplex(cos_half, 0)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_rx_gate(UINT target, const std::vector<double> &theta)
    {
        int32_t targets[] = {static_cast<int32_t>(target)};
        std::vector<int> matrix_indices(batch_size_);
        std::vector<hipDoubleComplex> matrices(batch_size_ * 4);

        for (int i = 0; i < batch_size_; i++) {
            double cos_half = std::cos(theta[i] / 2), sin_half = std::sin(theta[i] / 2);

            matrix_indices[i] = i;
            matrices[i * 4 + 0] = make_hipDoubleComplex(cos_half, 0);
            matrices[i * 4 + 1] = make_hipDoubleComplex(0, -sin_half);
            matrices[i * 4 + 2] = make_hipDoubleComplex(0, -sin_half);
            matrices[i * 4 + 3] = make_hipDoubleComplex(cos_half, 0);
        }

        HANDLE_ERROR(custatevecApplyMatrixBatched(
            handle_, state_, HIP_C_64F, n_, batch_size_, 1ULL << n_,
            CUSTATEVEC_MATRIX_MAP_TYPE_MATRIX_INDEXED, matrix_indices.data(), matrices.data(),
            HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, batch_size_, targets, 1, nullptr, nullptr,
            0, CUSTATEVEC_COMPUTE_64F, nullptr, 0));
    }

    void act_ry_gate(UINT target, double theta)
    {
        double cos_half = std::cos(theta / 2), sin_half = std::sin(theta / 2);
        hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(cos_half, 0), make_hipDoubleComplex(-sin_half, 0)},
            {make_hipDoubleComplex(sin_half, 0), make_hipDoubleComplex(cos_half, 0)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_ry_gate(UINT target, const std::vector<double> &theta)
    {
        int32_t targets[] = {static_cast<int32_t>(target)};
        std::vector<int> matrix_indices(batch_size_);
        std::vector<hipDoubleComplex> matrices(batch_size_ * 4);

        for (int i = 0; i < batch_size_; i++) {
            double cos_half = std::cos(theta[i] / 2), sin_half = std::sin(theta[i] / 2);

            matrix_indices[i] = i;
            matrices[i * 4 + 0] = make_hipDoubleComplex(cos_half, 0);
            matrices[i * 4 + 1] = make_hipDoubleComplex(-sin_half, 0);
            matrices[i * 4 + 2] = make_hipDoubleComplex(sin_half, 0);
            matrices[i * 4 + 3] = make_hipDoubleComplex(cos_half, 0);
        }

        HANDLE_ERROR(custatevecApplyMatrixBatched(
            handle_, state_, HIP_C_64F, n_, batch_size_, 1ULL << n_,
            CUSTATEVEC_MATRIX_MAP_TYPE_MATRIX_INDEXED, matrix_indices.data(), matrices.data(),
            HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, batch_size_, targets, 1, nullptr, nullptr,
            0, CUSTATEVEC_COMPUTE_64F, nullptr, 0));
    }

    void act_rz_gate(UINT target, double theta)
    {
        double cos_half = std::cos(theta / 2), sin_half = std::sin(theta / 2);
        hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(cos_half, -sin_half), make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(cos_half, sin_half)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_rz_gate(UINT target, const std::vector<double> &theta)
    {
        int32_t targets[] = {static_cast<int32_t>(target)};
        std::vector<int> matrix_indices(batch_size_);
        std::vector<hipDoubleComplex> matrices(batch_size_ * 4);

        for (int i = 0; i < batch_size_; i++) {
            double cos_half = std::cos(theta[i] / 2), sin_half = std::sin(theta[i] / 2);

            matrix_indices[i] = i;
            matrices[i * 4 + 0] = make_hipDoubleComplex(cos_half, -sin_half);
            matrices[i * 4 + 1] = make_hipDoubleComplex(0, 0);
            matrices[i * 4 + 2] = make_hipDoubleComplex(0, 0);
            matrices[i * 4 + 3] = make_hipDoubleComplex(cos_half, sin_half);
        }

        HANDLE_ERROR(custatevecApplyMatrixBatched(
            handle_, state_, HIP_C_64F, n_, batch_size_, 1ULL << n_,
            CUSTATEVEC_MATRIX_MAP_TYPE_MATRIX_INDEXED, matrix_indices.data(), matrices.data(),
            HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, batch_size_, targets, 1, nullptr, nullptr,
            0, CUSTATEVEC_COMPUTE_64F, nullptr, 0));
    }

    void act_p_gate(UINT target, double theta)
    {
        double cos = std::cos(theta), sin = std::sin(theta);
        hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(cos, sin)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_p_gate(UINT target, const std::vector<double> &theta)
    {
        int32_t targets[] = {static_cast<int32_t>(target)};
        std::vector<int> matrix_indices(batch_size_);
        std::vector<hipDoubleComplex> matrices(batch_size_ * 4);

        for (int i = 0; i < batch_size_; i++) {
            double cos = std::cos(theta[i] / 2), sin = std::sin(theta[i] / 2);

            matrix_indices[i] = i;
            matrices[i * 4 + 0] = make_hipDoubleComplex(1, 0);
            matrices[i * 4 + 1] = make_hipDoubleComplex(0, 0);
            matrices[i * 4 + 2] = make_hipDoubleComplex(0, 0);
            matrices[i * 4 + 3] = make_hipDoubleComplex(cos, sin);
        }

        HANDLE_ERROR(custatevecApplyMatrixBatched(
            handle_, state_, HIP_C_64F, n_, batch_size_, 1ULL << n_,
            CUSTATEVEC_MATRIX_MAP_TYPE_MATRIX_INDEXED, matrix_indices.data(), matrices.data(),
            HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, batch_size_, targets, 1, nullptr, nullptr,
            0, CUSTATEVEC_COMPUTE_64F, nullptr, 0));
    }

    void act_sx_gate(UINT target)
    {
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(0.5, 0.5), make_hipDoubleComplex(0.5, -0.5)},
            {make_hipDoubleComplex(0.5, -0.5), make_hipDoubleComplex(0.5, 0.5)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_sy_gate(UINT target)
    {
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(0.5, 0.5), make_hipDoubleComplex(-0.5, -0.5)},
            {make_hipDoubleComplex(0.5, 0.5), make_hipDoubleComplex(0.5, 0.5)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_sw_gate(UINT target)
    {
        static double inv_sqrt2 = 1 / std::sqrt(2);
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(inv_sqrt2, 0), make_hipDoubleComplex(-0.5, -0.5)},
            {make_hipDoubleComplex(0.5, -0.5), make_hipDoubleComplex(inv_sqrt2, 0)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_t_gate(UINT target)
    {
        static double inv_sqrt2 = 1 / std::sqrt(2);
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(inv_sqrt2, inv_sqrt2)}};

        act_single_qubit_gate(target, matrix);
    }

    void act_cx_gate(UINT control, UINT target)
    {
        static hipDoubleComplex matrix[4][4] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(1, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0),
             make_hipDoubleComplex(0, 0)},
        };

        act_two_qubit_gate(control, target, matrix);
    }

    void act_cz_gate(UINT control, UINT target)
    {
        static hipDoubleComplex matrix[4][4] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(-1, 0)},
        };

        act_two_qubit_gate(control, target, matrix);
    }

    void act_iswaplike_gate(UINT control, UINT target, double theta)
    {
        static hipDoubleComplex matrix[4][4] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(std::cos(theta), 0),
             make_hipDoubleComplex(0, -std::sin(theta)), make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, -std::sin(theta)),
             make_hipDoubleComplex(std::cos(theta), 0), make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(1, 0)},
        };

        act_two_qubit_gate(control, target, matrix);
    }

    void act_depolarizing_gate_1q(UINT target, double prob)
    {
        int32_t targets[] = {static_cast<int32_t>(target)};
        std::vector<int> matrix_indices(batch_size_);

        hipDoubleComplex matrices[] = {// I gate
                                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
                                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0),
                                      // X gate
                                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0),
                                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
                                      // Y gate
                                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, -1),
                                      make_hipDoubleComplex(0, 1), make_hipDoubleComplex(0, 0),
                                      // Z gate
                                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
                                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(-1, 0)};

        for (int sample = 0; sample < batch_size_; sample++) {
            double dice = dist_(mt_engine_);

            if (dice < prob / 3.0) {
                matrix_indices[sample] = 1;
            } else if (dice < prob * 2.0 / 3.0) {
                matrix_indices[sample] = 2;
            } else if (dice < prob) {
                matrix_indices[sample] = 3;
            }
        }

        HANDLE_ERROR(custatevecApplyMatrixBatched(
            handle_, state_, HIP_C_64F, n_, batch_size_, 1ULL << n_,
            CUSTATEVEC_MATRIX_MAP_TYPE_MATRIX_INDEXED, matrix_indices.data(), matrices, HIP_C_64F,
            CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, 4, targets, 1, nullptr, nullptr, 0,
            CUSTATEVEC_COMPUTE_64F, nullptr, 0));
    }

    void act_depolarizing_gate_2q(UINT control, UINT target, double prob)
    {
        act_depolarizing_gate_1q(control, 1.0 - std::sqrt(1.0 - prob));
        act_depolarizing_gate_1q(target, 1.0 - std::sqrt(1.0 - prob));
    }

    std::vector<std::complex<double>> observe(const Observable &obs) const
    {
        throw std::runtime_error("Not implemented");
    }

    void synchronize() { HANDLE_CUDA_ERROR(hipDeviceSynchronize()); }

private:
    static custatevecHandle_t handle_;
    hipDoubleComplex *state_;
    UINT batch_size_;
    UINT n_;

    std::random_device seed_gen_;
    std::mt19937 mt_engine_;
    std::uniform_real_distribution<double> dist_;
};

custatevecHandle_t State::Impl::handle_;

State::State(UINT n, UINT batch_size) : impl_(std::make_shared<Impl>(n, batch_size)) {}

State::~State() {}

void State::initialize() { Impl::initialize(); }

void State::finalize() { Impl::finalize(); }

std::vector<std::complex<double>> State::get_vector(UINT sample) const
{
    return impl_->get_vector(sample);
}

std::complex<double> State::amplitude(UINT sample, UINT basis) const
{
    return impl_->amplitude(sample, basis);
}

double State::re(UINT sample, UINT basis) const { return impl_->re(sample, basis); }

double State::im(UINT sample, UINT basis) const { return impl_->im(sample, basis); }

double State::get_probability(UINT basis) const { return impl_->get_probability(basis); }

double State::get_probability(UINT sample, UINT basis) const
{
    return impl_->get_probability(sample, basis);
}

std::vector<double> State::get_probability_batched(UINT basis) const
{
    return impl_->get_probability_batched(basis);
}

UINT State::dim() const { return impl_->dim(); }

UINT State::batch_size() const { return impl_->batch_size(); }

void State::set_zero_state() { return impl_->set_zero_state(); }

void State::act_x_gate(UINT target) { impl_->act_x_gate(target); }

void State::act_y_gate(UINT target) { impl_->act_x_gate(target); }

void State::act_z_gate(UINT target) { impl_->act_z_gate(target); }

void State::act_h_gate(UINT target) { impl_->act_h_gate(target); }

void State::act_rx_gate(UINT target, double theta) { impl_->act_rx_gate(target, theta); }

void State::act_rx_gate(UINT target, const std::vector<double> &theta)
{
    impl_->act_rx_gate(target, theta);
}

void State::act_ry_gate(UINT target, double theta) { impl_->act_ry_gate(target, theta); }

void State::act_ry_gate(UINT target, const std::vector<double> &theta)
{
    impl_->act_ry_gate(target, theta);
}

void State::act_rz_gate(UINT target, double theta) { impl_->act_rz_gate(target, theta); }

void State::act_rz_gate(UINT target, const std::vector<double> &theta)
{
    impl_->act_rz_gate(target, theta);
}

void State::act_p_gate(UINT target, double theta) { impl_->act_p_gate(target, theta); }

void State::act_p_gate(UINT target, const std::vector<double> &theta)
{
    impl_->act_p_gate(target, theta);
}

void State::act_sx_gate(UINT target) { impl_->act_sx_gate(target); }

void State::act_sy_gate(UINT target) { impl_->act_sy_gate(target); }

void State::act_sw_gate(UINT target) { impl_->act_sw_gate(target); }

void State::act_t_gate(UINT target) { impl_->act_t_gate(target); }

void State::act_cnot_gate(UINT control, UINT target) { impl_->act_cx_gate(control, target); }

void State::act_iswaplike_gate(UINT control, UINT target, double theta)
{

    impl_->act_iswaplike_gate(control, target, theta);
}

void State::act_cx_gate(UINT control, UINT target) { impl_->act_cx_gate(control, target); }

void State::act_cz_gate(UINT control, UINT target) { impl_->act_cz_gate(control, target); }

void State::act_depolarizing_gate_1q(UINT target, double prob)
{
    impl_->act_depolarizing_gate_1q(target, prob);
}

void State::act_depolarizing_gate_2q(UINT control, UINT target, double prob)
{
    impl_->act_depolarizing_gate_2q(control, target, prob);
}

std::vector<std::complex<double>> State::observe(const Observable &obs) const
{
    return impl_->observe(obs);
}

void State::synchronize() { impl_->synchronize(); }

void initialize() { State::initialize(); }

void finalize() { State::finalize(); }

} // namespace veqsim
