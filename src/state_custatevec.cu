#include "hip/hip_runtime.h"
#include <cmath>
#include <random>
#include <vector>

#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>
#include <custatevec.h>

#include "state.hpp"

#define HANDLE_ERROR(x)                                                                            \
    {                                                                                              \
        const auto err = x;                                                                        \
        if (err != CUSTATEVEC_STATUS_SUCCESS) {                                                    \
            printf("cuStateVec error \"%s\" at %s:%d\n", custatevecGetErrorString(err), __FILE__,  \
                   __LINE__);                                                                      \
            std::exit(EXIT_FAILURE);                                                               \
        }                                                                                          \
    };

#define HANDLE_CUDA_ERROR(x)                                                                       \
    {                                                                                              \
        const auto err = x;                                                                        \
        if (err != hipSuccess) {                                                                  \
            printf("CUDA Error: \"%s\" at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__);  \
            std::exit(EXIT_FAILURE);                                                               \
        }                                                                                          \
    };

class State::Impl
{
public:
    custatevecHandle_t handle_;
    hipDoubleComplex *state_;
    UINT batch_size_;
    UINT n_;

    std::random_device seed_gen_;
    std::mt19937 mt_engine_;
    std::uniform_real_distribution<double> dist_;

    Impl(UINT n, UINT batch_size)
        : n_(n), batch_size_(batch_size), mt_engine_(seed_gen_()), dist_(0.0, 1.0)
    {
        HANDLE_CUDA_ERROR(hipMalloc(&state_, batch_size * (1ULL << n) * sizeof(hipDoubleComplex)));
        HANDLE_ERROR(custatevecCreate(&handle_));
    }

    ~Impl()
    {
        HANDLE_CUDA_ERROR(hipFree(state_));
        HANDLE_ERROR(custatevecDestroy(handle_));
    }

    double re(UINT sample, UINT i)
    {
        hipDoubleComplex c;
        HANDLE_CUDA_ERROR(hipMemcpy(&c, state_ + (1ULL << n_) * sample + i,
                                     sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        return hipCreal(c);
    }

    double im(UINT sample, UINT i)
    {
        hipDoubleComplex c;
        HANDLE_CUDA_ERROR(hipMemcpy(&c, state_ + (1ULL << n_) * sample + i,
                                     sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
        return hipCimag(c);
    }

    double get_probability(UINT i)
    {
        // TODO
    }

    void set_zero_state()
    {
        std::vector<hipDoubleComplex> state(batch_size_ * (1ULL << n_));

        for (ITYPE i = 0; i < state.size(); i += (1ULL << n_)) {
            state[i] = make_hipDoubleComplex(1, 0);
        }

        HANDLE_CUDA_ERROR(hipMemcpy(state_, state.data(), state.size() * sizeof(hipDoubleComplex),
                                     hipMemcpyHostToDevice));
    }

    void act_single_qubit_gate(hipDoubleComplex matrix[2][2], UINT target)
    {
        int32_t targets[] = {static_cast<int32_t>(target)};

        // size_t workspace_size;
        // HANDLE_ERROR(custatevecApplyMatrixBatchedGetWorkspaceSize(
        //     handle_, HIP_C_64F, n_, batch_size_, (1ULL << n_),
        //     CUSTATEVEC_MATRIX_MAP_TYPE_BROADCAST, nullptr, matrix, HIP_C_64F,
        //     CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, 1, 1, 0, CUSTATEVEC_COMPUTE_64F,
        //     &workspace_size));

        HANDLE_ERROR(
            custatevecApplyMatrixBatched(handle_, state_, HIP_C_64F, n_, batch_size_, 1ULL << n_,
                                         CUSTATEVEC_MATRIX_MAP_TYPE_BROADCAST, nullptr, matrix,
                                         HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, 1, targets, 1,
                                         nullptr, nullptr, 0, CUSTATEVEC_COMPUTE_64F, nullptr, 0));
    }

    void act_two_qubit_gate(hipDoubleComplex matrix[4][4], UINT target, UINT control)
    {
        int32_t targets[] = {static_cast<int32_t>(target), static_cast<int32_t>(control)};

        // size_t workspace_size;
        // HANDLE_ERROR(custatevecApplyMatrixBatchedGetWorkspaceSize(
        //     handle_, HIP_C_64F, n_, batch_size_, (1ULL << n_),
        //     CUSTATEVEC_MATRIX_MAP_TYPE_BROADCAST, nullptr, matrix, HIP_C_64F,
        //     CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, 1, 2, 0, CUSTATEVEC_COMPUTE_64F,
        //     &workspace_size));

        HANDLE_ERROR(
            custatevecApplyMatrixBatched(handle_, state_, HIP_C_64F, n_, batch_size_, 1ULL << n_,
                                         CUSTATEVEC_MATRIX_MAP_TYPE_BROADCAST, nullptr, matrix,
                                         HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, 1, targets, 2,
                                         nullptr, nullptr, 0, CUSTATEVEC_COMPUTE_64F, nullptr, 0));
    }

    void act_x_gate(UINT target)
    {
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0)},
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0)}};

        act_single_qubit_gate(matrix, target);
    }

    void act_y_gate(UINT target)
    {
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, -1)},
            {make_hipDoubleComplex(0, -1), make_hipDoubleComplex(0, 0)}};

        act_single_qubit_gate(matrix, target);
    }

    void act_z_gate(UINT target)
    {
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(-1, 0)}};

        act_single_qubit_gate(matrix, target);
    }

    void act_h_gate(UINT target)
    {
        static double inv_sqrt2 = 1 / std::sqrt(2);
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(inv_sqrt2, 0), make_hipDoubleComplex(inv_sqrt2, 0)},
            {make_hipDoubleComplex(inv_sqrt2, 0), make_hipDoubleComplex(-inv_sqrt2, 0)}};

        act_single_qubit_gate(matrix, target);
    }

    void act_rx_gate(double theta, UINT target)
    {
        double cos_half = std::cos(theta / 2), sin_half = std::sin(theta / 2);
        hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(cos_half, 0), make_hipDoubleComplex(0, -sin_half)},
            {make_hipDoubleComplex(0, -sin_half), make_hipDoubleComplex(cos_half, 0)}};

        act_single_qubit_gate(matrix, target);
    }

    void act_ry_gate(double theta, UINT target)
    {
        double cos_half = std::cos(theta / 2), sin_half = std::sin(theta / 2);
        hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(cos_half, 0), make_hipDoubleComplex(-sin_half, 0)},
            {make_hipDoubleComplex(sin_half, 0), make_hipDoubleComplex(cos_half, 0)}};

        act_single_qubit_gate(matrix, target);
    }

    void act_rz_gate(double theta, UINT target)
    {
        double cos_half = std::cos(theta / 2), sin_half = std::sin(theta / 2);
        hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(cos_half, -sin_half), make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(cos_half, sin_half)}};

        act_single_qubit_gate(matrix, target);
    }

    void act_sx_gate(UINT target)
    {
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(0.5, 0.5), make_hipDoubleComplex(0.5, -0.5)},
            {make_hipDoubleComplex(0.5, -0.5), make_hipDoubleComplex(0.5, 0.5)}};

        act_single_qubit_gate(matrix, target);
    }

    void act_sy_gate(UINT target)
    {
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(0.5, 0.5), make_hipDoubleComplex(-0.5, -0.5)},
            {make_hipDoubleComplex(0.5, 0.5), make_hipDoubleComplex(0.5, 0.5)}};

        act_single_qubit_gate(matrix, target);
    }

    void act_sw_gate(UINT target)
    {
        static double inv_sqrt2 = 1 / std::sqrt(2);
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(inv_sqrt2, 0), make_hipDoubleComplex(-0.5, -0.5)},
            {make_hipDoubleComplex(0.5, -0.5), make_hipDoubleComplex(inv_sqrt2, 0)}};

        act_single_qubit_gate(matrix, target);
    }

    void act_t_gate(UINT target)
    {
        static double inv_sqrt2 = 1 / std::sqrt(2);
        static hipDoubleComplex matrix[2][2] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(inv_sqrt2, inv_sqrt2)}};

        act_single_qubit_gate(matrix, target);
    }

    void act_cnot_gate(UINT target, UINT control)
    {
        static hipDoubleComplex matrix[4][4] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(1, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0),
             make_hipDoubleComplex(0, 0)},
        };

        act_two_qubit_gate(matrix, target, control);
    }

    void act_cx_gate(UINT target, UINT control)
    {
        static hipDoubleComplex matrix[4][4] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(1, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
        };

        act_two_qubit_gate(matrix, target, control);
    }

    void act_cz_gate(UINT target, UINT control)
    {
        static hipDoubleComplex matrix[4][4] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(-1, 0)},
        };

        act_two_qubit_gate(matrix, target, control);
    }

    void act_iswaplike_gate(double theta, UINT target, UINT control)
    {
        static hipDoubleComplex matrix[4][4] = {
            {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(std::cos(theta), 0),
             make_hipDoubleComplex(0, -std::sin(theta)), make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, -std::sin(theta)),
             make_hipDoubleComplex(std::cos(theta), 0), make_hipDoubleComplex(0, 0)},
            {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
             make_hipDoubleComplex(1, 0)},
        };

        act_two_qubit_gate(matrix, target, control);
    }

    void act_depolarizing_gate_1q(UINT target, double prob)
    {
        int32_t targets[] = {static_cast<int32_t>(target)};
        std::vector<int> matrix_indices(batch_size_);

        hipDoubleComplex matrices[] = {// I gate
                                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
                                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0),
                                      // X gate
                                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0),
                                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
                                      // Y gate
                                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, -1),
                                      make_hipDoubleComplex(0, 1), make_hipDoubleComplex(0, 0),
                                      // Z gate
                                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
                                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(-1, 0)};

        for (int sample = 0; sample < batch_size_; sample++) {
            double dice = dist_(mt_engine_);

            if (dice < prob / 3.0) {
                matrix_indices[sample] = 1;
            } else if (dice < prob * 2.0 / 3.0) {
                matrix_indices[sample] = 2;
            } else if (dice < prob) {
                matrix_indices[sample] = 3;
            }
        }

        HANDLE_ERROR(custatevecApplyMatrixBatched(
            handle_, state_, HIP_C_64F, n_, batch_size_, 1ULL << n_,
            CUSTATEVEC_MATRIX_MAP_TYPE_MATRIX_INDEXED, matrix_indices.data(), matrices, HIP_C_64F,
            CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, 4, targets, 1, nullptr, nullptr, 0,
            CUSTATEVEC_COMPUTE_64F, nullptr, 0));
    }

    void act_depolarizing_gate_2q(UINT target, UINT control, double prob)
    {
        act_depolarizing_gate_1q(target, 1.0 - std::sqrt(1.0 - prob));
        act_depolarizing_gate_1q(control, 1.0 - std::sqrt(1.0 - prob));
    }

    void synchronize() { HANDLE_CUDA_ERROR(hipDeviceSynchronize()); }
};

State::State(UINT n, UINT batch_size) : impl_(std::make_shared<Impl>(n, batch_size)) {}

State::~State() {}

double State::re(UINT sample, UINT i) { return impl_->re(sample, i); }

double State::im(UINT sample, UINT i) { return impl_->im(sample, i); }

double State::get_probability(UINT i) { return impl_->get_probability(i); }

void State::set_zero_state() { return impl_->set_zero_state(); }

void State::act_x_gate(UINT target) { impl_->act_x_gate(target); }

void State::act_y_gate(UINT target) { impl_->act_x_gate(target); }

void State::act_z_gate(UINT target) { impl_->act_z_gate(target); }

void State::act_h_gate(UINT target) { impl_->act_h_gate(target); }

void State::act_rx_gate(double theta, UINT target) { impl_->act_rx_gate(theta, target); }

void State::act_ry_gate(double theta, UINT target) { impl_->act_ry_gate(theta, target); }

void State::act_rz_gate(double theta, UINT target) { impl_->act_rz_gate(theta, target); }

void State::act_sx_gate(UINT target) { impl_->act_sx_gate(target); }

void State::act_sy_gate(UINT target) { impl_->act_sy_gate(target); }

void State::act_sw_gate(UINT target) { impl_->act_sw_gate(target); }

void State::act_t_gate(UINT target) { impl_->act_t_gate(target); }

void State::act_cnot_gate(UINT target, UINT control) { impl_->act_cnot_gate(target, control); }

void State::act_iswaplike_gate(double theta, UINT target, UINT control)
{

    impl_->act_iswaplike_gate(theta, target, control);
}

void State::act_cx_gate(UINT target, UINT control) { impl_->act_cx_gate(target, control); }

void State::act_cz_gate(UINT target, UINT control) { impl_->act_cz_gate(target, control); }

void State::act_depolarizing_gate_1q(UINT target, double prob)
{
    impl_->act_depolarizing_gate_1q(target, prob);
}

void State::act_depolarizing_gate_2q(UINT target, UINT control, double prob)
{
    impl_->act_depolarizing_gate_2q(target, control, prob);
}

void State::synchronize() { impl_->synchronize(); }
